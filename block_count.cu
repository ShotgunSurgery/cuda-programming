
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printIDs(){
    printf("Block: %d, Thread: %d\n", blockIdx.x, threadIdx.x);
}

int main(){
    printIDs<<<3, 5>>>();
    hipDeviceSynchronize();
    return 0;

}

// x64 Native Tools Command Prompt for VS 2022 is a special terminal provided by vsc to compile 64 bit cpp code, we cannot 
// do it in vsc cause the libraries aren't in system path by default 

// CUDA uses SIMT (Single Instruction, Multiple Thread) model -> it picks blocks threads in accordance with the avalablity 
// of resources so they can be in any order, it decideds which block which wrap to exceute when 