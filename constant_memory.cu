#include "hip/hip_runtime.h"
// matrix multiplication using constant memory - 64KB usually, cached i.e. stores frequently used instructions, data enabling faster 
//retrival 

// each thread has threadIdx.x, threadIdx.y, threadIdx.z and they tell the position of the thread 
// within that block, hence if a block has 4*4 threads then threadIdx.x will be in range [0, 3] and so will 
// be threadIdx.y but threadIdx.z will be 0, so like for this thread -> (1, 2) -> threadIdx.x = 1 and threadIdx.y = 2

// the same as above also applies for blocks i.e. blockIdx.x ... 

// blockDim.x tell the number of threads in x dimension and so on ..
#include <stdio.h>

#define N 512

// the __constant__ keyword tells the compiler that this variable will recide in constant memory 
__constant__ float d_B[N][N];

__global__ void matrix_multiplication_constant(){

}   

int main(){
    // all these matrices would be of size N * N 
    float *host_matrix_a, *host_matrix_b, *host_matrix_c;
    float *device_matrix_a, *device_matrix_c;

    // size_t is unsigned int type 
    size_t bytes = N * N * sizeof(float);

    hipMemcpy(device_matrix_a, host_matrix_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_matrix_c, host_matrix_c, bytes, hipMemcpyHostToDevice);

    // the follwoing function is used to copy data from host to __constant__ variable
    // decond parameter is the source on host 
    hipMemcpyToSymbol(HIP_SYMBOL(d_B), host_matrix_b, N * N * sizeof(float));
    matrix_multiplication_constant<<<>>>(host_matrix_a, host_matrix_c, N);


         
}