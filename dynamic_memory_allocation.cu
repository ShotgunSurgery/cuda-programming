
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addTen(int* d_arr, int n){
    int idx = threadIdx.x;
    if(idx < n){
        d_arr[idx] += 10;
    }
}

int main(){
    int n = 7;
    int h_arr[] = {1,2,3,4,5,6,7};

    int* d_arr;

    // the job of the first parameter is to store the address of memory allocated on GPU, &d_arr is derefrencing it i.e. address of variable
    // d_arr, we need to change the value that the d_arr is pointing to so we need it's address which is stored in d_arr. that's why it's a pointer to a pointer 

    // Types of RAM -> 1. DRAM(Types -> 1. VRAM) 2. SRAM 
    // this memory is being allocated on the DRAM (Dynamic Random Access Memory) -> Nvidia RTX 4060 8GB here 8GB is refring to DRAM 
    hipMalloc( (void**)&d_arr, n * sizeof(int));


    // when a kernel is launched it's on the host memory(cpu) but for the gpu cores to access it we need to copy it on the GPU DRAM 
    // first param -> destination pointer memory location on gpu ram, second param -> source pointer memory location on host ram, third param -> number of bytes to copy, fourth 
    // param -> copy form host to gpu others can be cudaMemcpuyDeviceToHost() -> copy from Gpu to cpu 
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);

    addTen<<<1, n>>>(d_arr, n);

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < n; i++){
        printf("%d ",h_arr[i]);
    }
    printf("\n");

    // to free allocated memory on gpu 

    hipFree(d_arr);

    return 0;
}