
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addition_using_global_memory(int* host_arr1, int* host_arr2, int* result_arr){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < 7){
        result_arr[idx] = host_arr1[idx] + host_arr2[idx];
    }
}

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
hipEventCreate(&stop);

    int size = 7;
    int host_arr1[7] = {1,2,3,4,5,6,7};
    int host_arr2[7] = {1,2,3,4,5,6,7};
    int* result;
    int* device_arr1;
    int* device_arr2;
    // we are passing the address of the pointer device_arr1 to the function after typecasting it in void** cause it expects it that way
    // CUDA allocates the memory in VRAM and then store the address of that allocated memory in device_arr1, also it doesn't know what 
    // datatype we are going to store there so it just asks for void 
    hipMalloc((void**)& device_arr1, size * sizeof(int));
    hipMalloc((void**)& device_arr2,  size * sizeof(int));
    hipMalloc((void**)& result, size * sizeof(int));

    hipMemcpy(device_arr1, host_arr1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_arr2, host_arr2, sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start);
    addition_using_global_memory<<<1, 7>>>(device_arr1, device_arr2, result);

    hipDeviceSynchronize();
    hipEventRecord(stop);
hipEventSynchronize(stop);  // wait for event to complete

    int host_result[7];
hipMemcpy(host_result, result, size * sizeof(int), hipMemcpyDeviceToHost);

for(int j = 0; j < size; j++){
    printf("%d ", host_result[j]);
}

printf("\n");   
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
printf("Time taken by kernel: %f ms\n", milliseconds);

    hipFree(device_arr1);
    hipFree(device_arr2);
    hipFree(result);
    
}
