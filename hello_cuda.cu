// each cuda core runs multiple threads at a time

#include <hip/hip_runtime.h>
#include <iostream>

// function qualifier marks the function as kernel i.e. it will be called from the cpu but run on gpu 
// here kernel only means a function that runs on gpu 

/* hello_cuda.exe -> executable binary file
 hello_cuda.exp -> symbol export file
 hello_cuda.lib -> static import library */
__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

// when the program is compiled then the cpu and compiler are only working even when the program starts executing 
/* but when there is a 'kernel launch' i.e. the kernel is called only then will the cpu give the gpu the 
function to execute in parallel */
int main() {
    // <<how many blocks of threads to launch, threads per block>>(args);
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize(); // CUDA runtime api call which blocks the CPU until previously isshued GPU tasks aren't completed
    return 0;
}
