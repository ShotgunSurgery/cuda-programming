
#include <hip/hip_runtime.h>
#include <iostream>

// function qualifier marks the function as kernel i.e. it will be called from the cpu but run on gpu 
// hello_cuda.exe -> executable binary file
// hello_cuda.exp -> symbol export file
// hello_cuda.lib -> static import library
__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
