
#include <hip/hip_runtime.h>
#include <stdio.h>

// so when the first thread calls this function inside the threadIdx.x returns 1 and hence the 1st thread only works on 
// doubling one element 
__global__ void double_function(int* gpu_arr, int n){
    int idx = threadIdx.x;
    if(idx < n){
        gpu_arr[idx] *= 2;
    }
}
int main(){
    int n = 5;
    int cpu_arr[5] = {1,2,3,4,5};
    int* gpu_arr;
    hipMalloc( (void**)&gpu_arr, n * sizeof(int));
    hipMemcpy(gpu_arr, cpu_arr, n * sizeof(int), hipMemcpyHostToDevice);
    // when we write the following line then the gpu launches 1 block and n threads paralleley on it and each thread calls the kernel once
    // double_function<<<1, n>>>(gpu_arr, n);

    // an event is used to measure specific points in time during gpu execution 
    // below we declare two events 
    hipEvent_t start, stop;

float milliseconds = 0;

// here we "create" those events i.e. allocate memory to them and now these can record timestamps 
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);  // Start timing

double_function<<<1, n>>>(gpu_arr, n);

hipEventRecord(stop);   // Stop timing
    
hipEventSynchronize(stop);  // Wait for kernel to finish

hipEventElapsedTime(&milliseconds, start, stop);
printf("Kernel execution time: %f ms\n", milliseconds);

    hipMemcpy(cpu_arr, gpu_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < n; i++){
        printf("%d ", cpu_arr[i]);
    }
    printf("\n");   
    hipFree(gpu_arr);
    return 0;
}