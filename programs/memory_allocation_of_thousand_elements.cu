
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializer(int* gpu_arr){
    int thread_index = threadIdx.x;
    gpu_arr[thread_index] = 0;
}

int main(){
    int n = 1000;
    int* gpu_arr;
    int cpu_arr[1000];
    hipMalloc((void**)& gpu_arr, n * sizeof(int));
    // in one block only 1024 threads can be launched 
    initializer<<<1, 1000>>>(gpu_arr);
    hipMemcpy(cpu_arr, gpu_arr,10 * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < 10; i++){
        printf("%d ", cpu_arr[i]);
    }
    hipFree(gpu_arr);
    hipDeviceSynchronize();
}