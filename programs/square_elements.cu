
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square_fxn(int *arr, int n)
{
    int thread_index = threadIdx.x;
    // blockDim.x -> returns the number of threads in the block along x dimension
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        arr[i] *= arr[i];
    }
}

int main()
{
    int n = 4;
    int arr[4] = {1, 2, 3, 4};
    int *gpu_arr;
    hipMalloc((void **)&gpu_arr, n * sizeof(int));
    hipMemcpy(gpu_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    square_fxn<<<1, 2>>>(gpu_arr, n);
    hipMemcpy(arr, gpu_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < n; i++){
        printf("%d ", arr[i]);
    }
    hipFree(gpu_arr);
    return 0;
}