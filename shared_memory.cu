// program to add two arrays using shared memroy

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addition_using_shared_memory(int *a1, int *a2, int *out)
{
    __shared__ int a[7];
    __shared__ int b[7];
    // a1, a2 are pointer to memory
    // address in the global memory but we want to copy them into shared memory(on chip)
    // for more effecient working
    int idx = threadIdx.x;
    if (idx < 7)
    {
        a[idx] = a1[idx];
        b[idx] = a2[idx];
        out[idx] = a[idx] + b[idx];
    }
    __syncthreads();
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int arr1[7] = {1, 2, 3, 4, 5, 6, 7};
    int arr2[7] = {1, 2, 3, 4, 5, 6, 7};
    int *device_output;
    int *host_output;
    int size = 7;
    // we can never directly write into the VRAM so we always need to copy form the cpu 
    int *d_arr1;
    int *d_arr2;
    hipMalloc((void **)&d_arr1, size * sizeof(int));
    hipMalloc((void **)&d_arr2, size * sizeof(int));
    hipMemcpy(d_arr1, arr1, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, size * sizeof(int), hipMemcpyHostToDevice);
    
    host_output = (int *)malloc(size * sizeof(int));
    hipMalloc((void **)&device_output, size * sizeof(int));
    hipEventRecord(start);
    addition_using_shared_memory<<<1, size>>>(d_arr1, d_arr2, device_output);
    hipMemcpy(host_output, device_output, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    for (int j = 0; j < size; j++)
    {
        printf("%d ", host_output[j]);
    }
    printf("\n");
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);
    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(device_output);
    hipDeviceSynchronize();
    return 0;
}