#include <stdio.h>
#include <hip/hip_runtime.h> // CUDA runtime api -> provides runtime level functions and defs to 1. allocate memory
// 2. transfer data between CPU and GPU 3.Launch and manage kernels from host(CPU)

#define N 16 // preprocessor directive that defines a constant N with value 16, anywhere the code has N, it will be replaced with 16 before compilation