#include <stdio.h>
#include <hip/hip_runtime.h> // CUDA runtime api -> provides runtime level functions and defs to 1. allocate memory
// 2. transfer data between CPU and GPU 3.Launch and manage kernels from host(CPU)

#define N 16 // preprocessor directive that defines a constant N with value 16, anywhere the code has N, it will be replaced with 16 before compilation
#define BLOCK_SIZE 4

int main(){
    int h_input[N], h_output[N], i;

    for(i = 0; i < N; i++){
        h_input[i] = i;

        int *d_input, *d_output;
        
                    
    }
}