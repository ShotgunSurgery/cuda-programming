// shared memory -> avaliable to all threads in a block (but not between diffrent blocks) -> low latency/ size -> used 
// to temorarily store data in while the kernel execution 
// global memory -> GPU's VRAM -> avaliable to all blocks -> more latency/ size
#include <stdio.h>
#include <hip/hip_runtime.h> // CUDA runtime api -> provides runtime level functions and defs to 1. allocate memory
// 2. transfer data between CPU and GPU 3.Launch and manage kernels from host(CPU)

#define N 16 // preprocessor directive that defines a constant N with value 16, anywhere the code has N, it will be replaced with 16 before compilation
#define BLOCK_SIZE 4

__global__ void global_memory_kernel(int* device_input, int* device_output){
    // the following line calculates the global thread index 
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < N){
        device_output[index] = device_input[index] * 2;
    }
}

__global__ void shared_memory_kernel(int* input, int* output){
    // __shared__ is a CUDA storage qualifire, it marks the variable to be stored in shared memory
    __shared__ int temp[BLOCK_SIZE];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tx = threadIdx.x;

    // the input array in which we would be using 'idx' is of size 'N' so to prevent threads from accessing
    // memory beyond array size we check the following condition 
    if(idx < N){
        temp[tx] = input[idx];
        // __syncthreads() is a barrier syncronization function in CUDA -> makes all the threads in the block wait till 
        // everythread in the block has reached it 
        __syncthreads();
        
        output[idx]  = temp[tx] * 2;
    }
}

int main(){
    int host_input[N], host_output[N], i;
     int *device_input, *device_output;

    for(i = 0; i < N; i++){
        host_input[i] = i;

        hipMalloc((void**)& device_input, N * sizeof(int));
        hipMalloc((void**)& device_output, N * sizeof(int));

        hipMemcpy(device_input, host_input, N * sizeof(int), hipMemcpyHostToDevice);

        // grid is a collection of blocks 
        // dim3 is a CUDA specific datatype used to specify dimension of thread blocks or grid 
        dim3 blocks(N / BLOCK_SIZE); // this defines how many blocks will be launched in the grid 
        dim3 threads(BLOCK_SIZE); // this defines how many threads will be launched per block 

        printf("Running global memory kernel: \n");
        global_memory_kernel<<<blocks, threads>>>(device_input, device_output);
 hipMemcpy(host_output, device_output, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
        printf("%d ", host_output[i]);
    printf("\n");

    printf("Running shared memory kernel...\n");
    shared_memory_kernel<<<blocks, threads>>>(device_input, device_output);
    hipMemcpy(host_output, device_output, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
        printf("%d ", host_output[i]);
    printf("\n");

    hipFree(device_input);
    hipFree(device_output);


    }




}