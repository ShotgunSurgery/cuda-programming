
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void threadCount() {
    // threadIdx is a inbuilt variable that tells us about the id of the current thread 
    int id = threadIdx.x;
    printf("Thread %d\n", id);
}

int main(){
    // launch 1 block    with 5 threads 
    threadCount<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}

// Streamming Multiprocessor -> Runs multiple blocks 
// Grid -> group of blocks 
// Block -> group of threads
// Wrap -> group  of 32 threads